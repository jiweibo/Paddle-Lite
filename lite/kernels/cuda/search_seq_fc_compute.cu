// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "lite/backends/cuda/math/bias.h"
#include "lite/backends/cuda/math/type_trans.h"
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/search_seq_fc_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

template <typename T, PrecisionType PType>
void SearchSeqFcCompute<T, PType>::PrepareForRun() {
  gemm_impl_.reset(new lite::cuda::math::Gemm<T, T>);
}

template <typename T, PrecisionType PType>
void SearchSeqFcCompute<T, PType>::Run() {
  auto& param = this->template Param<param_t>();
  auto& cuda_ctx = this->ctx_->template As<HIPContext>();
  auto hip_stream = cuda_ctx.exec_stream();

  auto x = param.x;
  auto w = param.w;
  auto b = param.b;
  auto out = param.out;
  auto out_size = param.out_size;
  const auto x_dims = x->dims();
  const auto w_dims = w->dims();
  const auto out_dims = out->dims();
  CHECK_EQ(x_dims.size(), 2) << "The Input(X) should be 2-D tensor.";
  CHECK_EQ(w_dims.size(), 2) << "W should be 2-D tensor.";
  CHECK_EQ(out_dims.size(), 2) << "The Output(Out) should be 2-D tensor.";
  CHECK_EQ(x_dims[1], w_dims[1]) << "Wrong shape: x_dims[1] != w_dims[1]";
  CHECK_EQ(w_dims[0], out_size) << "Wrong shape: w_dims[0] != out_size";
  CHECK_EQ(out_dims[0], x_dims[0]) << "Wrong shape: out_dims[0] != x_dims[0]";
  CHECK_EQ(out_dims[1], out_size) << "Wrong shape: out_dims[1] != out_size";
  int M = x_dims[0];
  int K = x_dims[1];
  int N = w_dims[0];
  auto x_data = x->template data<T>();
  auto w_data = w->template data<T>();
  auto out_data = out->template mutable_data<T>(TARGET(kCUDA));

  CHECK(gemm_impl_->init(false, true, M, N, K, &cuda_ctx));
  gemm_impl_->run(1.0f, 0.0f, x_data, w_data, out_data, &cuda_ctx);

  if (b != nullptr) {
    auto b_dims = b->dims();
    CHECK_EQ(b_dims.size(), 1) << "b should be 1-D tensor.";
    CHECK_EQ(b_dims[0], w_dims[0]) << "Wrong shape: b_dims[0] != w_dims[0]";
    auto b_data = b->template data<T>();
    int total_size = M * N;
    lite::cuda::math::add_bias(total_size, N, b_data, out_data, hip_stream);
  }
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

using SeqFCFp32 =
    paddle::lite::kernels::cuda::SearchSeqFcCompute<float, PRECISION(kFloat)>;
using SeqFCFp16 =
    paddle::lite::kernels::cuda::SearchSeqFcCompute<half, PRECISION(kFP16)>;

REGISTER_LITE_KERNEL(search_seq_fc, kCUDA, kFloat, kNCHW, SeqFCFp32, def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("W", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("b", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();

REGISTER_LITE_KERNEL(search_seq_fc, kCUDA, kFP16, kNCHW, SeqFCFp16, def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindInput("W", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindInput("b", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .Finalize();
