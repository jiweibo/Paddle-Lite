// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "lite/backends/cuda/math/bias.h"
#include "lite/backends/cuda/math/type_trans.h"
#include "lite/core/op_registry.h"
#include "lite/kernels/cuda/search_seq_fc_compute.h"

namespace paddle {
namespace lite {
namespace kernels {
namespace cuda {

template <>
void SearchSeqFcCompute<float, PRECISION(kFloat)>::PrepareForRun() {
  gemm_impl_.reset(new lite::cuda::math::Gemm<float, float>);
  auto& param = this->Param<param_t>();
  w_tensor_ = param.w;
  b_tensor_ = param.b;
}

template <>
void SearchSeqFcCompute<half, PRECISION(kFP16)>::PrepareForRun() {
  gemm_impl_.reset(new lite::cuda::math::Gemm<half, half>);
  auto& param = this->Param<param_t>();
  w_half_tensor_.Resize(param.w->dims());
  lite::cuda::math::fp32_to_fp16(
      param.w->numel(),
      param.w->data<float>(),
      w_half_tensor_.mutable_data<half>(TARGET(kCUDA)));
  w_tensor_ = &w_half_tensor_;
  b_half_tensor_.Resize(param.b->dims());
  lite::cuda::math::fp32_to_fp16(
      param.b->numel(),
      param.b->data<float>(),
      b_half_tensor_.mutable_data<half>(TARGET(kCUDA)));
  b_half_tensor_.set_lod(param.b->lod());
  b_tensor_ = &b_half_tensor_;
}

template <typename T, PrecisionType PType>
void SearchSeqFcCompute<T, PType>::Run() {
  auto& param = this->template Param<param_t>();
  // CHECK(ctx_) << "running context should be set first";
  auto& cuda_ctx = this->ctx_->template As<HIPContext>();
  auto hip_stream = cuda_ctx.exec_stream();

  auto x = param.x;
  auto w = w_tensor_;
  auto b = b_tensor_;
  auto out = param.out;
  auto out_size = param.out_size;
  const auto x_dims = x->dims();
  const auto w_dims = w->dims();
  const auto out_dims = out->dims();
  CHECK_EQ(x_dims.size(), 2) << "The Input(X) should be 2-D tensor.";
  CHECK_EQ(w_dims.size(), 2) << "W should be 2-D tensor.";
  CHECK_EQ(out_dims.size(), 2) << "The Output(Out) should be 2-D tensor.";
  CHECK_EQ(x_dims[1], w_dims[1]) << "Wrong shape: x_dims[1] != w_dims[1]";
  CHECK_EQ(w_dims[0], out_size) << "Wrong shape: w_dims[0] != out_size";
  CHECK_EQ(out_dims[0], x_dims[0]) << "Wrong shape: out_dims[0] != x_dims[0]";
  CHECK_EQ(out_dims[1], out_size) << "Wrong shape: out_dims[1] != out_size";
  int M = x_dims[0];
  int K = x_dims[1];
  int N = w_dims[0];
  auto x_data = x->template data<T>();
  auto w_data = w->data<T>();
  auto out_data = out->template mutable_data<T>(TARGET(kCUDA));

  CHECK(gemm_impl_->init(false, true, M, N, K, &cuda_ctx));
  gemm_impl_->run(1.0f, 0.0f, x_data, w_data, out_data, &cuda_ctx);

  if (b != nullptr) {
    auto b_dims = b->dims();
    CHECK_EQ(b_dims.size(), 1) << "b should be 1-D tensor.";
    CHECK_EQ(b_dims[0], w_dims[0]) << "Wrong shape: b_dims[0] != w_dims[0]";
    auto b_data = b->data<T>();
    int total_size = M * N;
    lite::cuda::math::add_bias(total_size, N, b_data, out_data, hip_stream);
  }
}

}  // namespace cuda
}  // namespace kernels
}  // namespace lite
}  // namespace paddle

using SeqFCFp32 =
    paddle::lite::kernels::cuda::SearchSeqFcCompute<float, PRECISION(kFloat)>;
using SeqFCFp16 =
    paddle::lite::kernels::cuda::SearchSeqFcCompute<half, PRECISION(kFP16)>;

REGISTER_LITE_KERNEL(search_seq_fc, kCUDA, kFloat, kNCHW, SeqFCFp32, def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("W", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindInput("b", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA))})
    .Finalize();

REGISTER_LITE_KERNEL(search_seq_fc, kCUDA, kFP16, kNCHW, SeqFCFp16, def)
    .BindInput("X", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .BindInput("W", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFloat))})
    .BindInput("b", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFloat))})
    .BindOutput("Out", {LiteType::GetTensorTy(TARGET(kCUDA), PRECISION(kFP16))})
    .Finalize();
